#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : Avi_Lengthy_Multiplication_Table.cu
 Author      : Avimanyu Bandyopadhyay
 Email	     : avimanyu.bandyopadhyay@heritageit.edu.in
 Version     : 1.0
 Copyright   : Academic use only
 Description : CUDA Parallel Computation for Lengthy Multiplication Tables
 ============================================================================
 */



#include <iostream>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>	// Stops underlining of __global__ ; Also required for GPU elapsed time.

#include <>	// Stops underlining of threadIdx etc.

#define N (2048*2048)
#define THREADS_PER_BLOCK 512
#include <stdio.h>

// GPU kernel function to produce multiplication table of vectors 
__global__ void mult_table(unsigned long *a, unsigned long *b, unsigned long *c, unsigned long n){
	int index = threadIdx.x + blockIdx.x * blockDim.x; 
	if (index < n)
		c[index] = a[index] * b[index];
}


int main(void) {

	unsigned long *a, *b, *c; // host copies of a, b, c
	unsigned long *dev_a, *dev_b, *dev_c; // device copies of a, b, c
	unsigned long size = N * sizeof(unsigned long); // we need space for N integers
	unsigned long i, n;
	// Allocate GPU/device copies of dev_a, dev_b, dev_c
	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	// Allocate CPU/host copies of a, b, c
	a = (unsigned long *)malloc(size);
	b = (unsigned long *)malloc(size);
	c = (unsigned long *)malloc(size);

	printf("\nParallel Computation for Lengthy Multiplication Tables:");
	printf("\n-----------------------------------------------------");
	printf("\nMaximum Index: 4194304");
	printf("\nMaximum Product Limit: 4194304");
	printf("\n\nEnter a number for table:");
	scanf("%ld", &n);
	unsigned long n2 = n;
	// Setup input values
	for (i = 0; i < N - 1; ++i)
	{
		a[i] = n;
		b[i] = i;
	}

	// Copy inputs to device
	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

	//INITIALIZE CUDA EVENTS
	hipEvent_t start, stop;
	float elapsedTime; 

	//CREATING EVENTS
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//CUDA KERNEL STUFF HERE...
	// Launch mult_table() kernel on GPU with N threads
	mult_table << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(dev_a, dev_b, dev_c, N);

	//FINISH RECORDING
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	//CALCULATE ELAPSED TIME
	hipEventElapsedTime(&elapsedTime, start, stop);

	//DISPLAY COMPUTATION TIME WITH DEVICE NAME
	
	hipDeviceProp_t prop;
	int count;

	hipGetDeviceCount(&count);

	for (int igtx = 0; igtx < count; igtx++) {
		hipGetDeviceProperties(&prop, igtx);
		printf("\nGPU Device used for computation: %s\n", prop.name);
		printf("\nMultiplication Table for %ld computed in: %f milliseconds", n2,elapsedTime);
	}
	//printf("The GPU '%s' computed the multiplication table for %d in %f milliseconds", prop.name, n2, elapsedTime);
	//printf("\n\nGPU Computation Time = %f ms",elapsedTime);
	
	// Copy device result back to host copy of c
	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
	unsigned long begin, end;
		
	printf("\n\nChoose your indexes to display your required table:\n\n");

	printf("\nEnter initial index:");
	scanf("%ld", &begin);
	
	printf("\nEnter final index:");
	scanf("%ld", &end);
	printf("\n");
	for (i = begin; i <= end; ++i)
	{
		printf("%ld	X	%ld	=	%ld\n\n",n2, i, c[i]);
	}

	// Clean CPU memory allocations
	free(a); free(b); free(c);

	// Clean GPU memory allocations
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	//system("pause");
	return 0;
}

